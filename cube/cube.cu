
#include <hip/hip_runtime.h>
#include <stdio.h>

// 1. Note the convention d_* is used for device and h_* is used for host allocations.
// 2. __global__ tells cuda that what follows is a kernel implementation


// Cuda kernel that returns a cube of a given array
// Mostly written in a serial manner
__global__ void cube(float *d_out, float *d_in) {
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] = f * f * f;
}

int main(int argc, char **argv) {
  const int ARRAY_SIZE = 1000;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // Create input array on the host.
  float h_in[ARRAY_SIZE];
  for (int i=0; i < ARRAY_SIZE; i++) {
    h_in[i] = i;
  }
  float h_out[ARRAY_SIZE];

  // Declare pointers for GPU memory
  float *d_in;
  float *d_out;

  // Allocate memory on GPU.
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES);

  // Transfer array to GPU.
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  // Launch the kernel
  cube<<<1, ARRAY_SIZE>>>(d_out, d_in);

  // Copy back the result
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  // Print the resulting array
  for(int i=0; i < ARRAY_SIZE; i++) {
     printf("%f", h_out[i]);
     printf(((i % 4) != 3) ? "\t" : "\n");
  }

  hipFree(d_in);
  hipFree(d_out);

  return 0;
}

